#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void mulKernel(int *c, const int* a, const int* b,const int WIDTH){
  int x = threadIdx.x;
  int y = threadIdx.y;
  int i = y*(blockDim.x)+x;
  if(x<WIDTH && y<WIDTH){
    int sum =0;
  for(int k=0;k<WIDTH;k++){
    sum += a[y*WIDTH+k]*b[k*WIDTH+x];
  }
  c[i]=sum;
  }
}

int main(){
const int WIDTH = 5;
const int a[WIDTH][WIDTH]={1,2,3,4,5,6,7,8,9,10,1,2,3,4,5,6,7,8,9,10,1,1,1,1,1};
const int b[WIDTH][WIDTH]={2,3,4,5,6,7,8,9,10,1,2,3,4,5,6,7,8,9,10,1,2,3,4,5,6};
int c[WIDTH][WIDTH]={0};



int *dev_a, *dev_b,*dev_c=0;
hipMalloc((void**)&dev_a,WIDTH*WIDTH*sizeof(int));
hipMalloc((void**)&dev_b,WIDTH*WIDTH*sizeof(int));
hipMalloc((void**)&dev_c,WIDTH*WIDTH*sizeof(int));


hipMemcpy(dev_a,a,WIDTH*WIDTH*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_b,b,WIDTH*WIDTH*sizeof(int),hipMemcpyHostToDevice);

dim3 DimBlock(WIDTH,WIDTH);
mulKernel<<<1,DimBlock>>>(dev_c,dev_a,dev_b,WIDTH);
hipMemcpy(c,dev_c,WIDTH*WIDTH*sizeof(int),hipMemcpyDeviceToHost);

for(int y=0;y<WIDTH;y++){
  for(int x=0;x<WIDTH;x++){
    printf("%5d",c[y][x]);
  }
  printf("\n");
}

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}
