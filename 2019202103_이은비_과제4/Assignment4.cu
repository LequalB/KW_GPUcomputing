#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void MatrixMulKernel(int *c, const int* a, const int* b,const int WIDTH){
  int x = blockIdx.x*blockDim.x+threadIdx.x;
  int y = blockIdx.y*blockDim.y+threadIdx.y;
  int i = y*WIDTH+x;
  
  if(x<WIDTH && y<WIDTH){
    int sum =0;
  for(int k=0;k<WIDTH;k++){
    sum += a[y*WIDTH+k]*b[k*WIDTH+x];
  }
  c[i]=sum;
  }
}

int main(){
const int WIDTH = 16;
const int TILE_WIDTH = 8;
int a[WIDTH][WIDTH];
int b[WIDTH][WIDTH];
int c[WIDTH][WIDTH]={0};

for(int y=0;y<WIDTH;y++){
  for(int x=0;x<WIDTH;x++)
  {
    a[y][x]=x+y;
    b[y][x]=x+y;
  }

}


int *dev_a, *dev_b,*dev_c=0;
hipMalloc((void**)&dev_a,WIDTH*WIDTH*sizeof(int));
hipMalloc((void**)&dev_b,WIDTH*WIDTH*sizeof(int));
hipMalloc((void**)&dev_c,WIDTH*WIDTH*sizeof(int));


hipMemcpy(dev_a,a,WIDTH*WIDTH*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_b,b,WIDTH*WIDTH*sizeof(int),hipMemcpyHostToDevice);

dim3 dimGrid(WIDTH/TILE_WIDTH,WIDTH/TILE_WIDTH,1);
dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);

MatrixMulKernel<<<dimGrid,dimBlock>>>(dev_c,dev_a,dev_b,WIDTH);
hipMemcpy(c,dev_c,WIDTH*WIDTH*sizeof(int),hipMemcpyDeviceToHost);
for(int y=0;y<TILE_WIDTH;y++){
  
  for(int x=0;x<TILE_WIDTH;x++){
    printf("%5d",c[y][x]);
  }
  printf(" ");
  for(int x=TILE_WIDTH;x<WIDTH;x++){
    printf("%5d",c[y][x]); 
  }
  printf("\n");
  
}printf("\n");

for(int y=TILE_WIDTH;y<WIDTH;y++){
  
  for(int x=0;x<TILE_WIDTH;x++){
    printf("%5d",c[y][x]);
  }
  printf(" ");
  for(int x=TILE_WIDTH;x<WIDTH;x++){
    printf("%5d",c[y][x]); 
  }
  printf("\n");
  
}


  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}
