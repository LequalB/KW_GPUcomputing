#include <iostream>
#include <hip/hip_runtime.h>
__global__ void addKernel(int *d,const int *a,const int *b,const int *c){
    int i = threadIdx.x; //kernel function
    d[i]=a[i]+b[i]+c[i];
} //use PREFIX '__global__'that launched by CPU
int main(){
    const int SIZE = 5; //arraysize
    const int a[SIZE] = {1,2,3,4,5}; //initialize the values of source arrays
    const int b[SIZE] = {11,12,13,14,15};
    const int c[SIZE] = {21,22,23,24,25};
    int d[SIZE] ={0}; //initialize the array to store result value 
    
    int *dev_a = 0; //initialize variables
    int *dev_b = 0;
    int *dev_c = 0;
    int *dev_d = 0;

    hipMalloc((void**)&dev_d,SIZE*sizeof(int)); //Allocate the memory of the kernel 
    hipMalloc((void**)&dev_a,SIZE*sizeof(int));
    hipMalloc((void**)&dev_b,SIZE*sizeof(int));
    hipMalloc((void**)&dev_c,SIZE*sizeof(int));

    hipMemcpy(dev_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice); //Copy data from CPU to GPU
    hipMemcpy(dev_b,b,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_c,c,SIZE*sizeof(int),hipMemcpyHostToDevice);
    addKernel<<<1,SIZE>>>(dev_d,dev_a,dev_b,dev_c); //kernel function call
    hipDeviceSynchronize();

    hipMemcpy(d,dev_d,SIZE*sizeof(int),hipMemcpyDeviceToHost); //Copy data from GPU to CPU

    printf("{1,2,3,4,5}+{11,12,13,14,15}+{21,22,23,24,25} = {%d,%d,%d,%d,%d}\n",
           d[0],d[1],d[2],d[3],d[4]); //Print to check results

    hipFree(dev_d); //Release memory allocated to the kernel
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;

}
