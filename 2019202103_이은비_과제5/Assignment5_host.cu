
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <chrono>


#define GRIDSIZE 16*1024 //16K
#define BLOCKSIZE 1024 //1K
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE) //16M

void genData(float* ptr, unsigned int size) {
    while (size--) {
        *ptr++ = (float)(rand() % 1000) / 1000.0F;
    }
}

void adjDiff(float* dst, const float* src, unsigned int size) {
    for (int i = 1; i < size; i++) {
        dst[i] = src[i] - src[i - 1];
    }
}

int main() {
    float* pSource = NULL;
    float* pResult = NULL;
    int i;
    long long cntStart, cntEnd, freq = 0LL;

    pSource = (float*)malloc(TOTALSIZE * sizeof(float));
    pResult = (float*)malloc(TOTALSIZE * sizeof(float));
    //generate input source data
        genData(pSource, TOTALSIZE);
    //start the timer
        std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
    //calculate the adjacent difference
        pResult[0] = 0.0F; // exceptional case for i = 0
        adjDiff(pResult, pSource, TOTALSIZE);
    //end the timer
        std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
    std::chrono::nanoseconds duration_nano = end - start;
    printf("%lld\n", duration_nano);
    //print sample cases
        i = 1;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE - 1;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE / 2;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    free(pSource);
    free(pResult);
}
