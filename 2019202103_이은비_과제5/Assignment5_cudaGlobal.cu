
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/stat.h>
#include <chrono>


#define GRIDSIZE (16*1024) //16K
#define BLOCKSIZE 1024 //1K
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE) //16M

void genData(float* ptr, unsigned int size) {
    while (size--) {
        *ptr++ = (float)(rand() % 1000) / 1000.0F;
    }
}

__global__ void adjDiff(float* result, float* input) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i>0) {
        float x_i = input[i];
        float x_i_m1 = input[i-1];
        result[i] = x_i - x_i_m1;
    }
}

int main() {
    float* pSource = NULL;
    float* pResult = NULL;
    int i;
    long long cntStart, cntEnd, freq = 0LL;

    pSource = (float*)malloc(TOTALSIZE * sizeof(float));
    pResult = (float*)malloc(TOTALSIZE * sizeof(float));
    //generate input source data
        genData(pSource, TOTALSIZE);
        float* pSourceDev = NULL;
        float* pResultDev = NULL;
    //calculate the adjacent difference
        pResult[0] = 0.0F; // exceptional case for i = 0
        hipMalloc((void**)&pSourceDev,TOTALSIZE*sizeof(float));
        hipMalloc((void**)&pResultDev,TOTALSIZE*sizeof(float));
    //CUDA memcpy from host to device
      hipMemcpy(pSourceDev,pSource,TOTALSIZE*sizeof(float),hipMemcpyHostToDevice);
    //start the timer
        std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
   //CUDA launch the kernel adjDiff
   dim3 dimGrid(GRIDSIZE,1,1);
   dim3 dimBlock(BLOCKSIZE,1,1);
   adjDiff<<<dimGrid,dimBlock>>>(pResultDev,pSourceDev);
    //end the timer
        std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
    //CUDA memcpy from device to host
     hipMemcpy(pResult,pResultDev,TOTALSIZE*sizeof(float),hipMemcpyDeviceToHost);
    std::chrono::nanoseconds duration_nano = end - start;
    printf("%lld\n", duration_nano);
    //print sample cases
        i = 1;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE - 1;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE / 2;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    free(pSource);
    free(pResult);
    hipFree(pSourceDev);
    hipFree(pResultDev);
}
