
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/stat.h>
#include <chrono>


#define GRIDSIZE (16*1024) //16K
#define BLOCKSIZE 1024 //1K
#define TOTALSIZE (GRIDSIZE*BLOCKSIZE) //16M

void genData(float* ptr, unsigned int size) {
    while (size--) {
        *ptr++ = (float)(rand() % 1000) / 1000.0F;
    }
}

__global__ void adj_diff_shared(float* result,float* input){
  __shared__ float s_data[BLOCKSIZE];
  unsigned int tx = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  s_data[tx] = input[i];
  __syncthreads();
  if(tx>0){
    result[i] = s_data[tx] - s_data[tx-1];
  }
  else if(i>0){
    result[i] = s_data[tx]- input[i-1];
  }

}

int main() {
    float* pSource = NULL;
    float* pResult = NULL;
    int i;
    long long cntStart, cntEnd, freq = 0LL;

    pSource = (float*)malloc(TOTALSIZE * sizeof(float));
    pResult = (float*)malloc(TOTALSIZE * sizeof(float));
    //generate input source data
        genData(pSource, TOTALSIZE);
        float* pSourceDev = NULL;
        float* pResultDev = NULL;
    //calculate the adjacent difference
        pResult[0] = 0.0F; // exceptional case for i = 0
        hipMalloc((void**)&pSourceDev,TOTALSIZE*sizeof(float));
        hipMalloc((void**)&pResultDev,TOTALSIZE*sizeof(float));
    //CUDA memcpy from host to device
      hipMemcpy(pSourceDev,pSource,TOTALSIZE*sizeof(float),hipMemcpyHostToDevice);
    //start the timer
        std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
   //CUDA launch the kernel adjDiff
   dim3 dimGrid(GRIDSIZE,1,1);
   dim3 dimBlock(BLOCKSIZE,1,1);
   adj_diff_shared<<<dimGrid,dimBlock>>>(pResultDev,pSourceDev);
    //end the timer
        std::chrono::system_clock::time_point end = std::chrono::system_clock::now();
    //CUDA memcpy from device to host
     hipMemcpy(pResult,pResultDev,TOTALSIZE*sizeof(float),hipMemcpyDeviceToHost);
    std::chrono::nanoseconds duration_nano = end - start;
    printf("%lld\n", duration_nano);
    //print sample cases
        i = 1;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE - 1;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    i = TOTALSIZE / 2;
    printf("i=%7d : %f=%f-%f\n", i, pResult[i], pSource[i], pSource[i - 1]);
    free(pSource);
    free(pResult);
    hipFree(pSourceDev);
    hipFree(pResultDev);
}
